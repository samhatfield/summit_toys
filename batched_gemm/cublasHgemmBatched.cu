//
// Wrapper for cublasHgemm function. 
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h" 


bool alreadyAllocated_hgemm = false;
bool alreadyAllocated_hgemm_handle = false;

half **d_Aarray_hgemm;
half **d_Barray_hgemm;
half **d_Carray_hgemm;

half **Aarray_hgemm;
half **Barray_hgemm;
half **Carray_hgemm;

hipblasHandle_t handle_hgemm;	

extern "C" void cublasHgemmBatched_wrapper(
  char transa, char transb,
  int m, int n, int k,
  half alpha,
  const half *A, int lda, int tda,
  const half *B, int ldb, int tdb,
  half beta,
  half *C, int ldc, int tdc,
  int batchCount
){
  // Define CUBLAS operation handles
  hipblasOperation_t op_t1, op_t2;

  // Decide whether to transpose matrices or not
  op_t1 = (transa == 'T' || transa == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  op_t2 = (transb == 'T' || transb == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // Initialize CUBLAS handle
  if (!alreadyAllocated_hgemm_handle) {
    hipblasCreate(&handle_hgemm);
    alreadyAllocated_hgemm_handle = true;
  }

  // Allocate host arrays
  if (!alreadyAllocated_hgemm) {
    hipHostMalloc(&Aarray_hgemm, batchCount*sizeof(half*), hipHostMallocDefault);
    hipHostMalloc(&Barray_hgemm, batchCount*sizeof(half*), hipHostMallocDefault);
    hipHostMalloc(&Carray_hgemm, batchCount*sizeof(half*), hipHostMallocDefault);
    alreadyAllocated_hgemm = true;
  }

  // Allocate device arrays
  hipMalloc(&d_Aarray_hgemm, batchCount*sizeof(half*));
  hipMalloc(&d_Barray_hgemm, batchCount*sizeof(half*));
  hipMalloc(&d_Carray_hgemm, batchCount*sizeof(half*));

  // Transfer data from input arrays to host arrays
  for (int i = 0; i < batchCount; i++) {
    Aarray_hgemm[i] = (half*) &(A[i*lda*tda]);
    Barray_hgemm[i] = (half*) &(B[i*ldb*tdb]);
    Carray_hgemm[i] = (half*) &(C[i*ldc*tdc]);
  }

  // Transfer data from host arrays to device arrays
  hipMemcpy(d_Aarray_hgemm, Aarray_hgemm, batchCount*sizeof(half*), hipMemcpyHostToDevice);
  hipMemcpy(d_Barray_hgemm, Barray_hgemm, batchCount*sizeof(half*), hipMemcpyHostToDevice);
  hipMemcpy(d_Carray_hgemm, Carray_hgemm, batchCount*sizeof(half*), hipMemcpyHostToDevice);

  // Perform batched SGEMM
  hipblasGemmBatchedEx(handle_hgemm,
    op_t1, op_t2,
    m, n, k,
    (const void*)&alpha,
    (const void**)d_Aarray_hgemm, HIP_R_16F, lda,
    (const void**)d_Barray_hgemm, HIP_R_16F, ldb,
    (const void*)&beta,
    (void**)d_Carray_hgemm, HIP_R_16F, ldc,
    batchCount,
    HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);

  hipDeviceSynchronize();
  
  // Free device arrays
  hipFree(d_Aarray_hgemm);
  hipFree(d_Barray_hgemm);
  hipFree(d_Carray_hgemm);
}

extern "C" void cublasHgemmBatched_finalize ()
{

  if (alreadyAllocated_hgemm){
  
    hipFree(Aarray_hgemm);
    hipFree(Barray_hgemm);
    hipFree(Carray_hgemm);
    
    hipFree(d_Aarray_hgemm);
    hipFree(d_Barray_hgemm);
    hipFree(d_Carray_hgemm);

  }

  if (alreadyAllocated_hgemm_handle){
    hipblasDestroy(handle_hgemm);
  }
  
}
