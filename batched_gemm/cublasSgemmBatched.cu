//
// Wrapper for cublasSgemm function. 
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h" 


bool alreadyAllocated_sgemm = false;
bool alreadyAllocated_sgemm_handle = false;

float **d_Aarray_sgemm;
float **d_Barray_sgemm;
float **d_Carray_sgemm;

float **Aarray_sgemm;
float **Barray_sgemm;
float **Carray_sgemm;

hipblasHandle_t handle_sgemm;	

extern "C" void cublasSgemmBatched_wrapper(
  char transa, char transb,
  int m, int n, int k,
  float alpha,
  const float *A, int lda, int tda,
  const float *B, int ldb, int tdb,
  float beta,
  float *C, int ldc, int tdc,
  int batchCount
){
  // Define CUBLAS operation handles
  hipblasOperation_t op_t1, op_t2;

  // Decide whether to transpose matrices or not
  op_t1 = (transa == 'T' || transa == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  op_t2 = (transb == 'T' || transb == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // Initialize CUBLAS handle
  if (!alreadyAllocated_sgemm_handle) {
    hipblasCreate(&handle_sgemm);
    alreadyAllocated_sgemm_handle = true;
  }

  // Allocate host arrays
  if (!alreadyAllocated_sgemm) {
    hipHostMalloc(&Aarray_sgemm, batchCount*sizeof(float*), hipHostMallocDefault);
    hipHostMalloc(&Barray_sgemm, batchCount*sizeof(float*), hipHostMallocDefault);
    hipHostMalloc(&Carray_sgemm, batchCount*sizeof(float*), hipHostMallocDefault);
    alreadyAllocated_sgemm = true;
  }

  // Allocate device arrays
  hipMalloc(&d_Aarray_sgemm, batchCount*sizeof(float*));
  hipMalloc(&d_Barray_sgemm, batchCount*sizeof(float*));
  hipMalloc(&d_Carray_sgemm, batchCount*sizeof(float*));

  // Transfer data from input arrays to host arrays
  for (int i = 0; i < batchCount; i++) {
    Aarray_sgemm[i] = (float*) &(A[i*lda*tda]);
    Barray_sgemm[i] = (float*) &(B[i*ldb*tdb]);
    Carray_sgemm[i] = (float*) &(C[i*ldc*tdc]);
  }

  // Transfer data from host arrays to device arrays
  hipMemcpy(d_Aarray_sgemm,Aarray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);
  hipMemcpy(d_Barray_sgemm,Barray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);
  hipMemcpy(d_Carray_sgemm,Carray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);

  // Perform batched SGEMM
  hipblasGemmBatchedEx(handle_sgemm,
    op_t1, op_t2,
    m, n, k,
    (const void*)&alpha,
    (const void**)d_Aarray_sgemm, HIP_R_32F, lda,
    (const void**)d_Barray_sgemm, HIP_R_32F, ldb,
    (const void*)&beta,
    (void**)d_Carray_sgemm, HIP_R_32F, ldc,
    batchCount,
    HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

  hipDeviceSynchronize();
  
  // Free device arrays
  hipFree(d_Aarray_sgemm);
  hipFree(d_Barray_sgemm);
  hipFree(d_Carray_sgemm);
}

extern "C" void cublasSgemmBatched_finalize ()
{

  if (alreadyAllocated_sgemm){
  
    hipFree(Aarray_sgemm);
    hipFree(Barray_sgemm);
    hipFree(Carray_sgemm);
    
    hipFree(d_Aarray_sgemm);
    hipFree(d_Barray_sgemm);
    hipFree(d_Carray_sgemm);

  }

  if (alreadyAllocated_sgemm_handle){
    hipblasDestroy(handle_sgemm);
  }
  
}
